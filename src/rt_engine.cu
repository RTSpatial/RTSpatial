#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <optix_function_table_definition.h>  // for g_optixFunctionTable
#include <optix_host.h>
#include <optix_stack_size.h>
#include <optix_stubs.h>

#include <iostream>
#include <stdexcept>

#include "rtspatial/details/rt_engine.h"
#include "rtspatial/details/sbt_record.h"
#include "rtspatial/utils/exception.h"
#include "rtspatial/utils/util.h"

namespace rtspatial {
namespace details {
RTConfig get_default_rt_config(const std::string& ptx_root) {
  RTConfig config;

  {
    Module mod;

    mod.set_id(ModuleIdentifier::MODULE_ID_FLOAT_CONTAINS_POINT_QUERY_2D);
    mod.set_program_path(ptx_root +
                         "/float_shaders_contains_point_query_2d.ptx");
    mod.set_function_suffix("contains_point_query_2d");
    mod.EnableIsIntersection();
    mod.set_n_payload(1);

    config.AddModule(mod);

    mod.set_id(ModuleIdentifier::MODULE_ID_DOUBLE_CONTAINS_POINT_QUERY_2D);
    mod.set_program_path(ptx_root +
                         "/double_shaders_contains_point_query_2d.ptx");
    config.AddModule(mod);
  }

  {
    Module mod;

    mod.set_id(ModuleIdentifier::MODULE_ID_FLOAT_CONTAINS_ENVELOPE_QUERY_2D);
    mod.set_program_path(ptx_root +
                         "/float_shaders_contains_envelope_query_2d.ptx");
    mod.set_function_suffix("contains_envelope_query_2d");
    mod.EnableIsIntersection();
    mod.set_n_payload(1);

    config.AddModule(mod);

    mod.set_id(ModuleIdentifier::MODULE_ID_DOUBLE_CONTAINS_ENVELOPE_QUERY_2D);
    mod.set_program_path(ptx_root +
                         "/double_shaders_contains_envelope_query_2d.ptx");
    config.AddModule(mod);
  }

  {
    Module mod;

    mod.set_id(
        ModuleIdentifier::MODULE_ID_FLOAT_INTERSECTS_ENVELOPE_QUERY_2D_FORWARD);
    mod.set_program_path(ptx_root +
                         "/float_shaders_intersects_envelope_query_2d.ptx");
    mod.set_function_suffix("intersects_envelope_query_2d_forward");
    mod.EnableIsIntersection();
    mod.set_n_payload(2);

    config.AddModule(mod);

    mod.set_id(ModuleIdentifier::
                   MODULE_ID_DOUBLE_INTERSECTS_ENVELOPE_QUERY_2D_FORWARD);
    mod.set_program_path(ptx_root +
                         "/double_shaders_intersects_envelope_query_2d.ptx");
    config.AddModule(mod);

    mod.set_id(ModuleIdentifier::
                   MODULE_ID_FLOAT_INTERSECTS_ENVELOPE_QUERY_2D_BACKWARD);
    mod.set_program_path(ptx_root +
                         "/float_shaders_intersects_envelope_query_2d.ptx");
    mod.set_function_suffix("intersects_envelope_query_2d_backward");
    config.AddModule(mod);

    mod.set_id(ModuleIdentifier::
                   MODULE_ID_DOUBLE_INTERSECTS_ENVELOPE_QUERY_2D_BACKWARD);
    mod.set_program_path(ptx_root +
                         "/double_shaders_intersects_envelope_query_2d.ptx");
    config.AddModule(mod);
  }

  {
    Module mod;

    mod.set_id(
        ModuleIdentifier::MODULE_ID_FLOAT_CONTAINS_POINT_QUERY_2D_TRIANGLE);
    mod.set_program_path(ptx_root +
                         "/float_shaders_contains_point_query_2d_triangle.ptx");
    mod.set_function_suffix("contains_point_query_2d_triangle");
    mod.EnableAnyHit();
    mod.set_n_payload(1);

    config.AddModule(mod);

    mod.set_id(
        ModuleIdentifier::MODULE_ID_DOUBLE_CONTAINS_POINT_QUERY_2D_TRIANGLE);
    mod.set_program_path(
        ptx_root + "/double_shaders_contains_point_query_2d_triangle.ptx");
    config.AddModule(mod);
  }
#ifndef NDEBUG
  config.opt_level = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
  config.dbg_level = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
#else
  config.opt_level = OPTIX_COMPILE_OPTIMIZATION_LEVEL_3;
  config.dbg_level = OPTIX_COMPILE_DEBUG_LEVEL_NONE;
#endif

  return config;
}

void RTEngine::initOptix(const RTConfig& config) {
  // https://stackoverflow.com/questions/10415204/how-to-create-a-cuda-context
  hipFree(0);
  int numDevices;
  hipGetDeviceCount(&numDevices);
  if (numDevices == 0)
    throw std::runtime_error("#osc: no CUDA capable devices found!");

  // -------------------------------------------------------
  // initialize optix
  // -------------------------------------------------------
  OPTIX_CHECK(optixInit());
  h_launch_params_.resize(1024);
  launch_params_.resize(1024);
}

static void context_log_cb(unsigned int level, const char* tag,
                           const char* message, void*) {
  fprintf(stderr, "[%2d][%12s]: %s\n", (int) level, tag, message);
}

void RTEngine::createContext() {
  hipError_t cu_res = hipCtxGetCurrent(&cuda_context_);
  if (cu_res != hipSuccess)
    fprintf(stderr, "Error querying current context: error code %d\n", cu_res);
  OptixDeviceContextOptions options;
  options.logCallbackFunction = context_log_cb;
  options.logCallbackData = nullptr;

#ifndef NDEBUG
  options.logCallbackLevel = 4;
  options.validationMode = OptixDeviceContextValidationMode::
      OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL;
#else
  options.logCallbackLevel = 2;
#endif
  OPTIX_CHECK(
      optixDeviceContextCreate(cuda_context_, &options, &optix_context_));
}

void RTEngine::createModule(const RTConfig& config) {
  module_compile_options_.maxRegisterCount = config.max_reg_count;
  module_compile_options_.optLevel = config.opt_level;
  module_compile_options_.debugLevel = config.dbg_level;
  pipeline_compile_options_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  pipeline_link_options_.maxTraceDepth = config.max_trace_depth;

  modules_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  for (auto& pair : config.modules) {
    std::vector<char> programData = readData(pair.second.get_program_path());
    auto& pipeline_compile_options = pipeline_compile_options_[pair.first];

    //    pipeline_compile_options.traversableGraphFlags =
    //        OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
    pipeline_compile_options.traversableGraphFlags =
        OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_ANY;

    pipeline_compile_options.usesMotionBlur = false;
    pipeline_compile_options.numPayloadValues = pair.second.get_n_payload();
    pipeline_compile_options.numAttributeValues = pair.second.get_n_attribute();
    pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    pipeline_compile_options.pipelineLaunchParamsVariableName =
        RTSPATIAL_OPTIX_LAUNCH_PARAMS_NAME;
    //    pipeline_compile_options.usesPrimitiveTypeFlags =
    //        OPTIX_PRIMITIVE_TYPE_CUSTOM | OPTIX_PRIMITIVE_TYPE_SPHERE;

    char log[2048];
    size_t sizeof_log = sizeof(log);
    OPTIX_CHECK(optixModuleCreate(optix_context_, &module_compile_options_,
                                  &pipeline_compile_options, programData.data(),
                                  programData.size(), log, &sizeof_log,
                                  &modules_[pair.first]));
#ifndef NDEBUG
    if (sizeof_log > 1) {
      std::cout << log << std::endl;
    }
#endif
  }

  // External Module
  std::vector<char> programData =
      readData(config.external_module.get_program_path());
  auto& pipeline_compile_options =
      pipeline_compile_options_[ModuleIdentifier::MODULE_ID_EXTERNAL];

  //    pipeline_compile_options.traversableGraphFlags =
  //        OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
  pipeline_compile_options.traversableGraphFlags =
      OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_ANY;

  pipeline_compile_options.usesMotionBlur = false;
  pipeline_compile_options.numPayloadValues = 0;
  pipeline_compile_options.numAttributeValues = 0;
  pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
  pipeline_compile_options.pipelineLaunchParamsVariableName =
      RTSPATIAL_OPTIX_LAUNCH_PARAMS_NAME;

  char log[2048];
  size_t sizeof_log = sizeof(log);
  OPTIX_CHECK(optixModuleCreate(
      optix_context_, &module_compile_options_, &pipeline_compile_options,
      programData.data(), programData.size(), log, &sizeof_log,
      &modules_[ModuleIdentifier::MODULE_ID_EXTERNAL]));
#ifndef NDEBUG
  if (sizeof_log > 1) {
    std::cout << log << std::endl;
  }
#endif
}

void RTEngine::createExternalPrograms(const RTConfig& config) {
  OptixProgramGroupDesc pgd;
  OptixProgramGroupOptions pgOptions = {};

  auto func_name = std::string("__direct_callable__" +
                               config.external_module.get_function_suffix());

  pgd.kind = OPTIX_PROGRAM_GROUP_KIND_CALLABLES;
  pgd.flags = OPTIX_PROGRAM_GROUP_FLAGS_NONE;
  pgd.callables.moduleDC = modules_[ModuleIdentifier::MODULE_ID_EXTERNAL];
  pgd.callables.entryFunctionNameDC = func_name.c_str();
  pgd.callables.moduleCC = nullptr;
  pgd.callables.entryFunctionNameCC = nullptr;

  char log[2048];
  size_t sizeof_log = sizeof(log);
  OPTIX_CHECK(optixProgramGroupCreate(optix_context_, &pgd, 1, &pgOptions, log,
                                      &sizeof_log, &external_pg_));
  if (sizeof_log > 1) {
    std::cout << log << std::endl;
  }
}

void RTEngine::createRaygenPrograms(const RTConfig& config) {
  const auto& conf_modules = config.modules;
  raygen_pgs_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  for (auto& pair : conf_modules) {
    auto f_name = "__raygen__" + pair.second.get_function_suffix();
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
    pgDesc.raygen.module = modules_[pair.first];
    pgDesc.raygen.entryFunctionName = f_name.c_str();

    // OptixProgramGroup raypg;
    char log[2048];
    size_t sizeof_log = sizeof(log);
    OPTIX_CHECK(optixProgramGroupCreate(optix_context_, &pgDesc, 1, &pgOptions,
                                        log, &sizeof_log,
                                        &raygen_pgs_[pair.first]));
#ifndef NDEBUG
    if (sizeof_log > 1) {
      std::cout << log << std::endl;
    }
#endif
  }
}

/*! does all setup for the miss program(s) we are going to use */
void RTEngine::createMissPrograms(const RTConfig& config) {
  const auto& conf_modules = config.modules;
  miss_pgs_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  for (auto& pair : conf_modules) {
    auto& mod = pair.second;
    auto f_name = "__miss__" + mod.get_function_suffix();
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;

    pgDesc.miss.module = nullptr;
    pgDesc.miss.entryFunctionName = nullptr;

    if (mod.IsMissEnable()) {
      pgDesc.miss.module = modules_[pair.first];
      pgDesc.miss.entryFunctionName = f_name.c_str();
    }

    char log[2048];
    size_t sizeof_log = sizeof(log);
    OPTIX_CHECK(optixProgramGroupCreate(optix_context_, &pgDesc, 1, &pgOptions,
                                        log, &sizeof_log,
                                        &miss_pgs_[pair.first]));
#ifndef NDEBUG
    if (sizeof_log > 1) {
      std::cout << log << std::endl;
    }
#endif
  }
}

/*! does all setup for the hitgroup program(s) we are going to use */
void RTEngine::createHitgroupPrograms(const RTConfig& config) {
  auto& conf_modules = config.modules;
  hitgroup_pgs_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  for (auto& pair : conf_modules) {
    const auto& conf_mod = pair.second;
    auto f_name_anythit = "__anyhit__" + conf_mod.get_function_suffix();
    auto f_name_intersect = "__intersection__" + conf_mod.get_function_suffix();
    auto f_name_closesthit = "__closesthit__" + conf_mod.get_function_suffix();
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pg_desc = {};

    pg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;

    pg_desc.hitgroup.moduleIS = nullptr;
    pg_desc.hitgroup.entryFunctionNameIS = nullptr;
    pg_desc.hitgroup.moduleAH = nullptr;
    pg_desc.hitgroup.entryFunctionNameAH = nullptr;
    pg_desc.hitgroup.moduleCH = nullptr;
    pg_desc.hitgroup.entryFunctionNameCH = nullptr;

    if (conf_mod.IsIsIntersectionEnabled()) {
      pg_desc.hitgroup.moduleIS = modules_[pair.first];
      pg_desc.hitgroup.entryFunctionNameIS = f_name_intersect.c_str();
    }

    if (conf_mod.IsAnyHitEnable()) {
      pg_desc.hitgroup.moduleAH = modules_[pair.first];
      pg_desc.hitgroup.entryFunctionNameAH = f_name_anythit.c_str();
    }

    if (conf_mod.IsClosestHitEnable()) {
      pg_desc.hitgroup.moduleCH = modules_[pair.first];
      pg_desc.hitgroup.entryFunctionNameCH = f_name_closesthit.c_str();
    }

    char log[2048];
    size_t sizeof_log = sizeof(log);
    OPTIX_CHECK(optixProgramGroupCreate(optix_context_, &pg_desc, 1, &pgOptions,
                                        log, &sizeof_log,
                                        &hitgroup_pgs_[pair.first]));
#ifndef NDEBUG
    if (sizeof_log > 1) {
      std::cout << log << std::endl;
    }
#endif
  }
}

/*! assembles the full pipeline of all programs */
void RTEngine::createPipeline(const RTConfig& config) {
  pipelines_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  for (auto& pair : config.modules) {
    std::vector<OptixProgramGroup> program_groups;
    program_groups.push_back(raygen_pgs_[pair.first]);
    program_groups.push_back(miss_pgs_[pair.first]);
    program_groups.push_back(hitgroup_pgs_[pair.first]);
    program_groups.push_back(external_pg_);

    char log[2048];
    size_t sizeof_log = sizeof(log);
    OPTIX_CHECK(optixPipelineCreate(
        optix_context_, &pipeline_compile_options_[pair.first],
        &pipeline_link_options_, program_groups.data(),
        (int) program_groups.size(), log, &sizeof_log,
        &pipelines_[pair.first]));
#ifndef NDEBUG
    if (sizeof_log > 1) {
      std::cout << log << std::endl;
    }
#endif
    OptixStackSizes stack_sizes = {};
    for (auto& prog_group : program_groups) {
      OPTIX_CHECK(optixUtilAccumulateStackSizes(prog_group, &stack_sizes,
                                                pipelines_[pair.first]));
    }

    uint32_t direct_callable_stack_size_from_traversal;
    uint32_t direct_callable_stack_size_from_state;
    uint32_t continuation_stack_size;

    OPTIX_CHECK(optixUtilComputeStackSizes(
        &stack_sizes, config.max_trace_depth,
        0,  // maxCCDepth
        0,  // maxDCDepth
        &direct_callable_stack_size_from_traversal,
        &direct_callable_stack_size_from_state, &continuation_stack_size));
    OPTIX_CHECK(optixPipelineSetStackSize(
        pipelines_[pair.first], direct_callable_stack_size_from_traversal,
        direct_callable_stack_size_from_state, continuation_stack_size,
        config.max_traversable_depth  // maxTraversableDepth
        ));
  }
}

/*! constructs the shader binding table */
void RTEngine::buildSBT(const RTConfig& config) {
  sbts_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);
  raygen_records_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);
  miss_records_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);
  hitgroup_records_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);
  callable_records_.resize(ModuleIdentifier::NUM_MODULE_IDENTIFIERS);

  for (auto& pair : config.modules) {
    auto& sbt = sbts_[pair.first];
    std::vector<RaygenRecord> raygenRecords;
    {
      RaygenRecord rec;
      OPTIX_CHECK(optixSbtRecordPackHeader(raygen_pgs_[pair.first], &rec));
      rec.data = nullptr; /* for now ... */
      raygenRecords.push_back(rec);
    }
    raygen_records_[pair.first] = raygenRecords;
    sbt.raygenRecord = reinterpret_cast<hipDeviceptr_t>(
        thrust::raw_pointer_cast(raygen_records_[pair.first].data()));

    std::vector<MissRecord> missRecords;
    {
      MissRecord rec;
      OPTIX_CHECK(optixSbtRecordPackHeader(miss_pgs_[pair.first], &rec));
      rec.data = nullptr; /* for now ... */
      missRecords.push_back(rec);
    }

    miss_records_[pair.first] = missRecords;
    sbt.missRecordBase = reinterpret_cast<hipDeviceptr_t>(
        thrust::raw_pointer_cast(miss_records_[pair.first].data()));
    sbt.missRecordStrideInBytes = sizeof(MissRecord);
    sbt.missRecordCount = (int) missRecords.size();
    sbt.callablesRecordBase = 0;

    std::vector<HitgroupRecord> hitgroupRecords;
    {
      HitgroupRecord rec;
      OPTIX_CHECK(optixSbtRecordPackHeader(hitgroup_pgs_[pair.first], &rec));
      rec.data = nullptr;
      hitgroupRecords.push_back(rec);
    }
    hitgroup_records_[pair.first] = hitgroupRecords;
    sbt.hitgroupRecordBase = reinterpret_cast<hipDeviceptr_t>(
        thrust::raw_pointer_cast(hitgroup_records_[pair.first].data()));
    sbt.hitgroupRecordStrideInBytes = sizeof(HitgroupRecord);
    sbt.hitgroupRecordCount = (int) hitgroupRecords.size();

    std::vector<CallableRecord> callableRecords;
    {
      CallableRecord rec;
      OPTIX_CHECK(optixSbtRecordPackHeader(external_pg_, &rec));
      rec.data = nullptr;
      callableRecords.push_back(rec);
    }
    callable_records_[pair.first] = callableRecords;
    sbt.callablesRecordBase = reinterpret_cast<hipDeviceptr_t>(
        thrust::raw_pointer_cast(callable_records_[pair.first].data()));
    sbt.callablesRecordStrideInBytes = sizeof(CallableRecord);
    sbt.callablesRecordCount = (int) callableRecords.size();
  }
}

OptixTraversableHandle RTEngine::buildAccel(hipStream_t hip_stream,
                                            ArrayView<OptixAabb> aabbs,
                                            ReusableBuffer& buf,
                                            bool prefer_fast_build) {
  OptixTraversableHandle traversable;
  OptixBuildInput build_input = {};
  hipDeviceptr_t d_aabb = THRUST_TO_CUPTR(aabbs.data());
  // Setup AABB build input. Don't disable AH.
  uint32_t build_input_flags[1] = {OPTIX_GEOMETRY_FLAG_NONE};
  uint32_t num_prims = aabbs.size();

  assert(reinterpret_cast<uint64_t>(aabbs.data()) %
             OPTIX_AABB_BUFFER_BYTE_ALIGNMENT ==
         0);

  build_input.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
  build_input.customPrimitiveArray.aabbBuffers = &d_aabb;
  build_input.customPrimitiveArray.flags = build_input_flags;
  build_input.customPrimitiveArray.numSbtRecords = 1;
  build_input.customPrimitiveArray.numPrimitives = num_prims;
  // it's important to pass 0 to sbtIndexOffsetBuffer
  build_input.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
  build_input.customPrimitiveArray.sbtIndexOffsetSizeInBytes = sizeof(uint32_t);
  build_input.customPrimitiveArray.primitiveIndexOffset = 0;

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  // FIXME: prefer_fast_build is consistent with updateAccel
  if (prefer_fast_build) {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
  } else {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
  }
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));

  // Alignment
  buf.Acquire(getAccelAlignedSize(buf.GetTail()) - buf.GetTail());
  char* out_buf = buf.Acquire(blas_buffer_sizes.outputSizeInBytes);
  char* temp_buf = buf.Acquire(blas_buffer_sizes.tempSizeInBytes);

  OPTIX_CHECK(optixAccelBuild(
      optix_context_, hip_stream, &accelOptions, &build_input, 1,
      reinterpret_cast<hipDeviceptr_t>(temp_buf),
      blas_buffer_sizes.tempSizeInBytes, reinterpret_cast<hipDeviceptr_t>(out_buf),
      blas_buffer_sizes.outputSizeInBytes, &traversable, nullptr, 0));

  buf.Release(blas_buffer_sizes.tempSizeInBytes);
  return traversable;
}

OptixTraversableHandle RTEngine::updateAccel(hipStream_t hip_stream,
                                             OptixTraversableHandle handle,
                                             ArrayView<OptixAabb> aabbs,
                                             ReusableBuffer& buf,
                                             size_t buf_offset,
                                             bool prefer_fast_build) {
  OptixBuildInput build_input = {};
  hipDeviceptr_t d_aabb = THRUST_TO_CUPTR(aabbs.data());
  // Setup AABB build input. Don't disable AH.
  // OPTIX_GEOMETRY_FLAG_DISABLE_ANYHIT
  uint32_t build_input_flags[1] = {OPTIX_GEOMETRY_FLAG_NONE};
  uint32_t num_prims = aabbs.size();

  assert(reinterpret_cast<uint64_t>(aabbs.data()) %
             OPTIX_AABB_BUFFER_BYTE_ALIGNMENT ==
         0);

  build_input.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
  build_input.customPrimitiveArray.aabbBuffers = &d_aabb;
  build_input.customPrimitiveArray.flags = build_input_flags;
  build_input.customPrimitiveArray.numSbtRecords = 1;
  build_input.customPrimitiveArray.numPrimitives = num_prims;
  // it's important to pass 0 to sbtIndexOffsetBuffer
  build_input.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
  build_input.customPrimitiveArray.sbtIndexOffsetSizeInBytes = sizeof(uint32_t);
  build_input.customPrimitiveArray.primitiveIndexOffset = 0;

  // ==================================================================
  // Bottom-level acceleration structure (BLAS) setup
  // ==================================================================

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_UPDATE;
  if (prefer_fast_build) {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
  } else {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
  }

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));
  char* out_buf = buf.GetData() + buf_offset;
  size_t tail = buf.GetTail();
  // Alignment
  buf.Acquire(getAccelAlignedSize(buf.GetTail()) - buf.GetTail());
  char* temp_buf = buf.Acquire(blas_buffer_sizes.tempUpdateSizeInBytes);

  OPTIX_CHECK(optixAccelBuild(
      optix_context_, hip_stream, &accelOptions, &build_input, 1,
      THRUST_TO_CUPTR(temp_buf), blas_buffer_sizes.tempUpdateSizeInBytes,
      THRUST_TO_CUPTR(out_buf), blas_buffer_sizes.outputSizeInBytes, &handle,
      nullptr, 0));
  buf.SetTail(tail);
  return handle;
}

OptixTraversableHandle RTEngine::buildAccelTriangle(hipStream_t hip_stream,
                                                    ArrayView<float3> vertices,
                                                    ArrayView<uint3> indices,
                                                    ReusableBuffer& buf,
                                                    bool prefer_fast_build) {
  OptixTraversableHandle traversable;
  OptixBuildInput build_input = {};
  auto d_indices = reinterpret_cast<hipDeviceptr_t>(indices.data());
  auto d_vertices = reinterpret_cast<hipDeviceptr_t>(vertices.data());
  // Setup AABB build input. Don't disable AH.
  uint32_t build_input_flags[1] = {
      OPTIX_GEOMETRY_FLAG_NONE |
      OPTIX_GEOMETRY_FLAG_REQUIRE_SINGLE_ANYHIT_CALL};

  memset(&build_input.triangleArray, 0, sizeof(build_input.triangleArray));

  build_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
  build_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
  build_input.triangleArray.vertexStrideInBytes = sizeof(float3);
  build_input.triangleArray.numVertices = vertices.size();
  build_input.triangleArray.vertexBuffers = &d_vertices;

  if (indices.empty()) {
    build_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_NONE;
  } else {
    build_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
    build_input.triangleArray.indexStrideInBytes = sizeof(uint3);
    build_input.triangleArray.numIndexTriplets = indices.size();
    build_input.triangleArray.indexBuffer = d_indices;
  }
  build_input.triangleArray.flags = build_input_flags;
  build_input.triangleArray.numSbtRecords = 1;
  // it's important to pass 0 to sbtIndexOffsetBuffer
  build_input.triangleArray.sbtIndexOffsetBuffer = 0;
  build_input.triangleArray.sbtIndexOffsetSizeInBytes = 0;
  build_input.triangleArray.primitiveIndexOffset = 0;

  // ==================================================================
  // Bottom-level acceleration structure (BLAS) setup
  // ==================================================================

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  if (prefer_fast_build) {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
  } else {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
  }
#ifndef NDEBUG
  accelOptions.buildFlags |= OPTIX_BUILD_FLAG_ALLOW_RANDOM_VERTEX_ACCESS;
#endif
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));
  // Alignment
  buf.Acquire(getAccelAlignedSize(buf.GetTail()) - buf.GetTail());
  char* out_buf = buf.Acquire(blas_buffer_sizes.outputSizeInBytes);
  char* temp_buf = buf.Acquire(blas_buffer_sizes.tempSizeInBytes);

  OPTIX_CHECK(optixAccelBuild(
      optix_context_, hip_stream, &accelOptions, &build_input, 1,
      THRUST_TO_CUPTR(temp_buf), blas_buffer_sizes.tempSizeInBytes,
      THRUST_TO_CUPTR(out_buf), blas_buffer_sizes.outputSizeInBytes,
      &traversable, nullptr, 0));

  buf.Release(blas_buffer_sizes.tempSizeInBytes);

  return traversable;
}

OptixTraversableHandle RTEngine::updateAccelTriangle(hipStream_t hip_stream,
                                                     ArrayView<float3> vertices,
                                                     ArrayView<uint3> indices,
                                                     ReusableBuffer& buf,
                                                     size_t buf_offset,
                                                     bool prefer_fast_build) {
  OptixTraversableHandle traversable;
  OptixBuildInput build_input = {};
  auto d_indices = reinterpret_cast<hipDeviceptr_t>(indices.data());
  auto d_vertices = reinterpret_cast<hipDeviceptr_t>(vertices.data());
  // Setup AABB build input. Don't disable AH.
  uint32_t build_input_flags[1] = {
      OPTIX_GEOMETRY_FLAG_NONE |
      OPTIX_GEOMETRY_FLAG_REQUIRE_SINGLE_ANYHIT_CALL};

  memset(&build_input.triangleArray, 0, sizeof(build_input.triangleArray));

  build_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
  build_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
  build_input.triangleArray.vertexStrideInBytes = sizeof(float3);
  build_input.triangleArray.numVertices = vertices.size();
  build_input.triangleArray.vertexBuffers = &d_vertices;

  if (indices.empty()) {
    build_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_NONE;
  } else {
    build_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
    build_input.triangleArray.indexStrideInBytes = sizeof(uint3);
    build_input.triangleArray.numIndexTriplets = indices.size();
    build_input.triangleArray.indexBuffer = d_indices;
  }
  build_input.triangleArray.flags = build_input_flags;
  build_input.triangleArray.numSbtRecords = 1;
  // it's important to pass 0 to sbtIndexOffsetBuffer
  build_input.triangleArray.sbtIndexOffsetBuffer = 0;
  build_input.triangleArray.sbtIndexOffsetSizeInBytes = 0;
  build_input.triangleArray.primitiveIndexOffset = 0;

  // ==================================================================
  // Bottom-level acceleration structure (BLAS) setup
  // ==================================================================

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  if (prefer_fast_build) {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
  } else {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
  }
#ifndef NDEBUG
  accelOptions.buildFlags |= OPTIX_BUILD_FLAG_ALLOW_RANDOM_VERTEX_ACCESS;
#endif
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_UPDATE;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));
  char* out_buf = buf.GetData() + buf_offset;
  size_t tail = buf.GetTail();
  // Alignment
  buf.Acquire(getAccelAlignedSize(buf.GetTail()) - buf.GetTail());
  char* temp_buf = buf.Acquire(blas_buffer_sizes.tempUpdateSizeInBytes);

  OPTIX_CHECK(optixAccelBuild(
      optix_context_, hip_stream, &accelOptions, &build_input, 1,
      THRUST_TO_CUPTR(temp_buf), blas_buffer_sizes.tempSizeInBytes,
      THRUST_TO_CUPTR(out_buf), blas_buffer_sizes.outputSizeInBytes,
      &traversable, nullptr, 0));

  buf.SetTail(tail);

  return traversable;
}

OptixTraversableHandle RTEngine::buildInstanceAccel(
    hipStream_t hip_stream, ArrayView<OptixInstance> instances,
    ReusableBuffer& buf, bool prefer_fast_build) {
  OptixTraversableHandle traversable;
  OptixBuildInput build_input = {};

  build_input.type = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
  build_input.instanceArray.instances =
      reinterpret_cast<hipDeviceptr_t>(instances.data());
  build_input.instanceArray.numInstances = instances.size();

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  if (prefer_fast_build) {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
  } else {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
  }
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));
  // Alignment
  buf.Acquire(getAccelAlignedSize(buf.GetTail()) - buf.GetTail());
  char* out_buf = buf.Acquire(blas_buffer_sizes.outputSizeInBytes);
  char* temp_buf = buf.Acquire(blas_buffer_sizes.tempSizeInBytes);

  OPTIX_CHECK(optixAccelBuild(
      optix_context_, hip_stream, &accelOptions, &build_input, 1,
      THRUST_TO_CUPTR(temp_buf), blas_buffer_sizes.tempSizeInBytes,
      THRUST_TO_CUPTR(out_buf), blas_buffer_sizes.outputSizeInBytes,
      &traversable, nullptr, 0));

  buf.Release(blas_buffer_sizes.tempSizeInBytes);
  return traversable;
}

OptixTraversableHandle RTEngine::updateInstanceAccel(
    hipStream_t hip_stream, ArrayView<OptixInstance> instances,
    ReusableBuffer& buf, size_t buf_offset, bool prefer_fast_build) {
  OptixTraversableHandle traversable;
  OptixBuildInput build_input = {};

  build_input.type = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
  build_input.instanceArray.instances =
      reinterpret_cast<hipDeviceptr_t>(instances.data());
  build_input.instanceArray.numInstances = instances.size();

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  if (prefer_fast_build) {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_BUILD;
  } else {
    accelOptions.buildFlags |= OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
  }
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));

  char* out_buf = buf.GetData() + buf_offset;
  size_t tail = buf.GetTail();
  // Alignment
  buf.Acquire(getAccelAlignedSize(buf.GetTail()) - buf.GetTail());
  char* temp_buf = buf.Acquire(blas_buffer_sizes.tempUpdateSizeInBytes);

  OPTIX_CHECK(optixAccelBuild(
      optix_context_, hip_stream, &accelOptions, &build_input, 1,
      THRUST_TO_CUPTR(temp_buf), blas_buffer_sizes.tempSizeInBytes,
      THRUST_TO_CUPTR(out_buf), blas_buffer_sizes.outputSizeInBytes,
      &traversable, nullptr, 0));

  buf.SetTail(tail);

  return traversable;
}

void RTEngine::Render(hipStream_t hip_stream, ModuleIdentifier mod,
                      dim3 dim) {
  void* launch_params = thrust::raw_pointer_cast(launch_params_.data());

  OPTIX_CHECK(optixLaunch(/*! pipeline we're launching launch: */
                          pipelines_[mod], hip_stream,
                          /*! parameters and SBT */
                          reinterpret_cast<hipDeviceptr_t>(launch_params),
                          params_size_, &sbts_[mod],
                          /*! dimensions of the launch: */
                          dim.x, dim.y, dim.z));
}

size_t RTEngine::EstimateMemoryUsageForAABB(size_t num_aabbs) {
  OptixBuildInput build_input = {};
  uint32_t build_input_flags[1] = {OPTIX_GEOMETRY_FLAG_NONE};

  build_input.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
  build_input.customPrimitiveArray.aabbBuffers = nullptr;
  build_input.customPrimitiveArray.flags = build_input_flags;
  build_input.customPrimitiveArray.numSbtRecords = 1;
  build_input.customPrimitiveArray.numPrimitives = num_aabbs;
  // it's important to pass 0 to sbtIndexOffsetBuffer
  build_input.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
  build_input.customPrimitiveArray.sbtIndexOffsetSizeInBytes = sizeof(uint32_t);
  build_input.customPrimitiveArray.primitiveIndexOffset = 0;

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));
  return blas_buffer_sizes.outputSizeInBytes +
         blas_buffer_sizes.tempSizeInBytes;
}

size_t RTEngine::EstimateMemoryUsageForTriangle(size_t num_aabb) {
  uint32_t build_input_flags[1] = {
      OPTIX_GEOMETRY_FLAG_NONE |
      OPTIX_GEOMETRY_FLAG_REQUIRE_SINGLE_ANYHIT_CALL};
  OptixBuildInput build_input = {};
  memset(&build_input.triangleArray, 0, sizeof(build_input.triangleArray));

  build_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
  build_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
  build_input.triangleArray.vertexStrideInBytes = sizeof(float3);
  build_input.triangleArray.numVertices = 4 * num_aabb;

  build_input.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
  build_input.triangleArray.indexStrideInBytes = sizeof(uint3);
  build_input.triangleArray.numIndexTriplets = 2 * num_aabb;
  build_input.triangleArray.flags = build_input_flags;
  build_input.triangleArray.numSbtRecords = 1;
  // it's important to pass 0 to sbtIndexOffsetBuffer
  build_input.triangleArray.sbtIndexOffsetBuffer = 0;
  build_input.triangleArray.sbtIndexOffsetSizeInBytes = 0;
  build_input.triangleArray.primitiveIndexOffset = 0;

  OptixAccelBuildOptions accelOptions = {};
  accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_UPDATE;
  accelOptions.motionOptions.numKeys = 1;
  accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

  OptixAccelBufferSizes blas_buffer_sizes;
  OPTIX_CHECK(optixAccelComputeMemoryUsage(optix_context_, &accelOptions,
                                           &build_input,
                                           1,  // num_build_inputs
                                           &blas_buffer_sizes));
  return blas_buffer_sizes.outputSizeInBytes +
         blas_buffer_sizes.tempSizeInBytes;
}
}  // namespace details
}  // namespace rtspatial