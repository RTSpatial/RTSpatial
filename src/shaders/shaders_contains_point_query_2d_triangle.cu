#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <optix.h>
#include <optix_device.h>

#include "rtspatial/details/launch_parameters.h"

enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };
// FLOAT_TYPE is defined by CMakeLists.txt
extern "C" __constant__
    rtspatial::details::LaunchParamsContainsPoint<FLOAT_TYPE, 2>
        params;

extern "C" __global__ void __anyhit__contains_point_query_2d_triangle() {
  auto primitive_idx = optixGetPrimitiveIndex();
  auto box_id = primitive_idx / 2;
  auto inst_id = optixGetInstanceId();
  auto geom_id = params.prefix_sum[inst_id] + box_id;
  auto query_id = optixGetPayload_0();
  const auto& envelope = params.envelopes[geom_id];
  const auto& query = params.queries[query_id];
  const auto& min_corner = envelope.get_min();
  const auto& max_corner = envelope.get_max();

  //  if (envelope.Contains(query)) {
  rtspatial_handle_point_contains(geom_id, query_id, params.arg);
  //  }
  optixIgnoreIntersection();
}

extern "C" __global__ void __raygen__contains_point_query_2d_triangle() {
  const auto& queries = params.queries;
  float tmin = 0;
  float tmax = 1;

  for (auto i = optixGetLaunchIndex().x; i < queries.size();
       i += optixGetLaunchDimensions().x) {
    const auto& p = queries[i];
    float3 origin;
    origin.x = p.get_x();
    origin.y = p.get_y();
    origin.z = -0.1;

    float3 dir = {0, 0, 1};

    optixTrace(params.handle, origin, dir, tmin, tmax, 0,
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,     // SBT offset
               RAY_TYPE_COUNT,       // SBT stride
               SURFACE_RAY_TYPE,     // missSBTIndex
               i);
  }
}
